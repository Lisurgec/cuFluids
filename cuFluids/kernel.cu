#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Point.h"
#include "Heap.h"
#include "KDTree.h"
#include "Box.h"

#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	KDTree k;
	/*Point3D p = Point3D(1, 1, 1);
	Point3D q = Point3D(2, 2, 2);
	Point3D r = Point3D(3, 3, 3);

	k.insert(&p);
	k.insert(&q);
	k.insert(&r);*/

	for (int i = 0; i < 10; i++)
	{
		Point3D p = Point3D(i, i, i);
		k.insert(&p);
	}
	


	std::vector <Point3D> a = k.flatten();
	for (auto b : a)
	{
		std::cout << b << std::endl;
	}

	system("PAUSE");

}
